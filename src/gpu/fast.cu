#include "hip/hip_runtime.h"
#include "gpu/fast.h"

#include <iostream>
#include <stdio.h>

__device__
int position(int m,int n,int width)
{
  int idx=m+n*width;
  return idx;
}
__global__
void fast(uchar* image, int width, int height,Corner* d_corner,int gridsize_x, int gridsize_y, const int threshold)
{
  __shared__ uchar patch[22][22];
  uint sp=0,sn=0;
  int m=blockDim.x*blockIdx.x+threadIdx.x;
  int n=blockDim.y*blockIdx.y+threadIdx.y;
  uint idx = m+n*width;
  uint idx_block=threadIdx.y*blockDim.x+threadIdx.x;               //offset for pixel in patch
	
  (d_corner+idx)->set=0;
  (d_corner+idx)->value=0;
  int patch_top_x=blockDim.x*blockIdx.x-3;
  int patch_top_y=blockDim.y*blockIdx.y-3;
  int idx_block_256=idx_block+256;

  //load into patch
  patch[idx_block%22][idx_block/22]=image[position(patch_top_x+idx_block%22,patch_top_y+idx_block/22,width)];
  if(idx_block_256<484)
    patch[(idx_block+256)%22][(idx_block+256)/22]=image[position(patch_top_x+idx_block_256%22,patch_top_y+idx_block_256/22,width)];
  __syncthreads();

  //detect
  int x=3+threadIdx.x;
  int y=3+threadIdx.y;
  if(m>2&&m<(width-3)&&n>2&&n<(height-3))            //detect the points away from the edges
    {
      uchar center_value = patch[x][y];
      sp |=(patch[x][y-3]>(center_value + threshold))<< 0;
      sp |=(patch[x+1][y-3]>(center_value + threshold))<< 1;
      sp |=(patch[x+2][y-2]>(center_value + threshold))<< 2;
      sp |=(patch[x+3][y-1]>(center_value + threshold))<< 3;
      sp |=(patch[x+3][y]>(center_value + threshold))<< 4;
      sp |=(patch[x+3][y+1]>(center_value + threshold))<< 5;
      sp |=(patch[x+2][y+2]>(center_value + threshold))<< 6;
      sp |=(patch[x+1][y+3]>(center_value + threshold))<< 7;
      sp |=(patch[x][y+3]>(center_value + threshold))<< 8;
      sp |=(patch[x-1][y+3]>(center_value + threshold))<< 9;
      sp |=(patch[x-2][y+2]>(center_value + threshold))<< 10;
      sp |=(patch[x-3][y+1]>(center_value + threshold))<< 11;
      sp |=(patch[x-3][y]>(center_value + threshold))<< 12;
      sp |=(patch[x-3][y-1]>(center_value + threshold))<< 13;
      sp |=(patch[x-2][y-2]>(center_value + threshold))<< 14;
      sp |=(patch[x-1][y-3]>(center_value + threshold))<< 15;

      sp+=sp<<16;
      uint sp1=sp&(sp<<1);
      uint sp2=sp1&(sp1<<2);
      uint sp3=sp2&(sp2<<4);
      uint sp4=sp3&(sp<<8);
      if(sp4!=0)
	{
	  int value=abs(center_value-patch[x-1][y-1])+abs(center_value-patch[x][y-1])+abs(center_value-patch[x+1][y-1])+
	    abs(center_value-patch[x-1][y])+abs(center_value-patch[x+1][y])+abs(center_value-patch[x+1][y-1])+
	    abs(center_value-patch[x+1][y])+abs(center_value-patch[x+1][y+1]);
	  d_corner[idx].value=value;
	  d_corner[idx].set=1;
	}
      else
	{
	  sn |=(patch[x][y-3]<(center_value - threshold))<< 0;
	  sn |=(patch[x+1][y-3]<(center_value - threshold))<< 1;
	  sn |=(patch[x+2][y-2]<(center_value - threshold))<< 2;
	  sn |=(patch[x+3][y-1]<(center_value - threshold))<< 3;
	  sn |=(patch[x+3][y]<(center_value - threshold))<< 4;
	  sn |=(patch[x+3][y+1]<(center_value - threshold))<< 5;
	  sn |=(patch[x+2][y+2]<(center_value - threshold))<< 6;
	  sn |=(patch[x+1][y+3]<(center_value - threshold))<< 7;
	  sn |=(patch[x][y+3]>(center_value - threshold))<< 8;
	  sn |=(patch[x-1][y+3]<(center_value - threshold))<< 9;
	  sn |=(patch[x-2][y+2]<(center_value - threshold))<< 10;
	  sn |=(patch[x-3][y+1]<(center_value - threshold))<< 11;
	  sn |=(patch[x-3][y]<(center_value - threshold))<< 12;
	  sn |=(patch[x-3][y-1]<(center_value - threshold))<< 13;
	  sn |=(patch[x-2][y-2]<(center_value - threshold))<< 14;
	  sn |=(patch[x-1][y-3]<(center_value - threshold))<< 15;
	  sn+=sn<<16;
	  uint sn1=sn&(sn<<1);
	  uint sn2=sn1&(sn1<<2);
	  uint sn3=sn2&(sn2<<4);
	  uint sn4=sn3&(sn<<8);
	  if(sn4!=0)
	    {
	      int value=abs(center_value-patch[x-1][y-1])+abs(center_value-patch[x][y-1])+abs(center_value-patch[x+1][y-1])+
		abs(center_value-patch[x-1][y])+abs(center_value-patch[x+1][y])+abs(center_value-patch[x+1][y-1])+
		abs(center_value-patch[x+1][y])+abs(center_value-patch[x+1][y+1]);
	      d_corner[idx].value=value;
	      d_corner[idx].set=1;
	      printf("");
	    }
	}
    }

}
__global__
void nms(uchar* image, Corner* d_corner,int width, int height)
{
  int m=blockDim.x*blockIdx.x+threadIdx.x;
  int n=blockDim.y*blockIdx.y+threadIdx.y;
  int idx=n*width+m;
  if(d_corner[idx].set==1)
    {
      int corner_value=d_corner[idx].value;
      if(d_corner[position(m-1,n-1,width)].value> corner_value)
	{d_corner[idx].set=0;return;}
      if(d_corner[position(m,n-1,width)].value> corner_value)
	{d_corner[idx].set=0;return;}
      if(d_corner[position(m+1,n-1,width)].value> corner_value)
	{d_corner[idx].set=0;return;}
      if(d_corner[position(m-1,n,width)].value> corner_value)
	{d_corner[idx].set=0;return;}
      if(d_corner[position(m+1,n,width)].value> corner_value)
	{d_corner[idx].set=0;return;}
      if(d_corner[position(m+1,n-1,width)].value> corner_value)
	{d_corner[idx].set=0;return;}
      if(d_corner[position(m+1,n,width)].value> corner_value)
	{d_corner[idx].set=0;return;}
      if(d_corner[position(m+1,n+1,width)].value> corner_value)
	{d_corner[idx].set=0;return;}

    }
}

GpuFast::GpuFast()
{


}

GpuFast::~GpuFast()
{


}

void GpuFast::create(int width,int height)
{
  m_rows = height;
  m_cols = width;
  hipStreamCreate(&m_stream);
  hipMallocManaged( &m_pCorner,sizeof(Corner)*m_rows*m_cols);
  hipMallocManaged( &m_pData, sizeof(unsigned char)*m_rows*m_cols);
  //checkCudaErrors(hipMallocManaged( &m_pCorner,sizeof(Corner)*m_rows*m_cols));
  //checkCudaErrors(hipMallocManaged( &m_pData, sizeof(bunsigned char)*m_rows*m_cols));
}

void GpuFast::detect(cv::Mat image, std::vector<cv::KeyPoint>& keyPoints, int threshold,bool nonmaxSuppression)
{
  dim3 blocksize(16,16);
  dim3 gridsize((m_cols-1)/blocksize.x+1, (m_rows-1)/blocksize.y+1);

  int rows = image.rows;
  int cols = image.cols;
  //
  memcpy(m_pData, image.data, sizeof(unsigned char)*rows*cols);

  //kenerl
  fast<<<gridsize,blocksize,0,m_stream>>>(m_pData, cols, rows,m_pCorner,gridsize.x,gridsize.y,threshold);
  nms<<<gridsize,blocksize,0,m_stream>>>(m_pData,m_pCorner,cols,rows);
  hipStreamSynchronize(m_stream);

  for(int i=0;i<cols*rows;i++) {
    if((m_pCorner+i)->set != 0) { 
      int x=i%cols;	 
      int y=i/cols;
      //KeyPoint (float x,float y,float _size,float _angle=-1,float _response=0,int _octave=0,int _class_id=-1)	
      cv::KeyPoint kpt((float)x,(float)y,0);
      kpt.response = (m_pCorner+i)->value;
      keyPoints.push_back(kpt);     
    }
  }
}

void GpuFast::destroy()
{
  hipFree(m_pData);
  hipFree(m_pCorner);
  hipStreamDestroy(m_stream);
}
